#include "hip/hip_runtime.h"
/* Each kernel handles the update of one pagerank score. In other
 * words, each kernel handles one row of the update:
 *
 *      pi(t+1) = (1/2) A pi(t) + (1 / (2N))
 *      
 * You may assume that num_nodes <= blockDim.x * 65535
 *
 */
#include<stdio.h>
__global__
void device_graph_propagate(const uint* graph_indices
		, const uint* graph_edges
		, const float* graph_nodes_in
		, float* graph_nodes_out
		, const float* inv_edges_per_node
		, int num_nodes) {
	// TODO: fill in the kernel code here
	int k= threadIdx.x + blockDim.x * blockIdx.x;
	if(k<num_nodes){
//	printf("--%d-/%d-\n",k,num_nodes);
		float sum=0.f;
		for(uint j=graph_indices[k];j<graph_indices[k+1] ; j++){
			sum+= graph_nodes_in[ graph_edges[j] ] * inv_edges_per_node[graph_edges[j] ];
		}
		graph_nodes_out[k] = 0.5f/(float)num_nodes + 0.5f*sum;
	}
}

/* This function executes a specified number of iterations of the
 * pagerank algorithm. The variables are:
 *
 * h_graph_indices, h_graph_edges:
 *     These arrays describe the indices of the neighbors of node i.
 *     Specifically, node i is adjacent to all nodes in the range
 *     h_graph_edges[h_graph_indices[i] ... h_graph_indices[i+1]].
 *
 * h_node_values_input:
 *     An initial guess of pi(0).
 *
 * h_gpu_node_values_output:
 *     Output array for the pagerank vector.
 *
 * h_inv_edges_per_node:
 *     The i'th element in this array is the reciprocal of the
 *     out degree of the i'th node.
 *
 * nr_iterations:
 *     The number of iterations to run the pagerank algorithm for.
 *
 * num_nodes:
 *     The number of nodes in the whole graph (ie N).
 *
 * avg_edges:
 *     The average number of edges in the graph. You are guaranteed
 *     that the whole graph has num_nodes * avg_edges edges.
 *
 */
double device_graph_iterate(const uint* h_graph_indices
		, const uint* h_graph_edges
		, const float* h_node_values_input
		, float* h_gpu_node_values_output
		, const float* h_inv_edges_per_node
		, int nr_iterations
		, int num_nodes
		, int avg_edges) {
	// TODO: allocate GPU memory
	float* buffer_1, *buffer_2;
	uint* graph_indices,* graph_edges;
	float * inv_edges_per_node;
	if(hipMalloc((void**)&buffer_1,num_nodes*sizeof(float))==-1){
		check_launch("gpu allocate failure");
	}
	if(hipMalloc((void**)&buffer_2,num_nodes*sizeof(float))==-1){
		check_launch("gpu allocate failure");
	}
	if(hipMalloc((void**)&graph_indices,(num_nodes+1)*sizeof(uint))==-1){
		check_launch("gpu allocate failure");
	}
	if(hipMalloc((void**)&graph_edges,(num_nodes*avg_edges)*sizeof(uint))==-1){
		check_launch("gpu allocate failure");
	}
	if(hipMalloc((void**)&inv_edges_per_node,num_nodes*sizeof(float))==-1){
		check_launch("gpu allocate failure");
	}

	// TODO: check for allocation failure

	// TODO: copy data to the GPU
	hipMemcpy(buffer_1,h_node_values_input,num_nodes*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(graph_indices,h_graph_indices,(num_nodes+1)*sizeof(uint),hipMemcpyHostToDevice);
	hipMemcpy(graph_edges,h_graph_edges,(num_nodes*avg_edges)*sizeof(uint),hipMemcpyHostToDevice);
	hipMemcpy(inv_edges_per_node,h_inv_edges_per_node,num_nodes*sizeof(float),hipMemcpyHostToDevice);
	start_timer(&timer);

	const int block_size = 1024;

	// TODO: launch your kernels the appropriate number of iterations
	for(int iter=0;iter<nr_iterations / 2 ; iter++){
		device_graph_propagate<<<num_nodes/block_size+1,block_size>>>(graph_indices,graph_edges,buffer_1,buffer_2,inv_edges_per_node,num_nodes);
		device_graph_propagate<<<num_nodes/block_size+1,block_size>>>(graph_indices,graph_edges,buffer_2,buffer_1,inv_edges_per_node,num_nodes);
	}

	check_launch("gpu graph propagate");
//		printf("\n------------------------gpu start---------------------------\n\n");
	double gpu_elapsed_time = stop_timer(&timer);

	// TODO: copy final data back to the host for correctness checking
	if(nr_iterations %2){
		device_graph_propagate<<<num_nodes/block_size+1,block_size>>>(graph_indices,graph_edges,buffer_1,buffer_2,inv_edges_per_node,num_nodes);
		hipMemcpy(h_gpu_node_values_output, buffer_2, num_nodes*sizeof(float), hipMemcpyDeviceToHost);
	}else{
		hipMemcpy(h_gpu_node_values_output, buffer_1, num_nodes*sizeof(float), hipMemcpyDeviceToHost);
	}
	// TODO: free the memory you allocated!
	hipFree(buffer_1);
	hipFree(buffer_2);
	hipFree(graph_indices);
	hipFree(graph_edges);
	hipFree(inv_edges_per_node);
	return gpu_elapsed_time;
}
